#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

// We have to know if there is a runtime error
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void compute_next_generation(int *d_grid, int *d_new_grid, int N) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // column
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    int idx = y * N + x; // cell index
	// initialize neighbors counter
    int live_neighbors = 0;
	// Ensure thread is within the bounds of grid
	if (x < N && y < N){
	//printf("_Cell_num-%d,%d,%d-_\n", idx, x, y);
	
		// Calculate the number of live neighbors
		for (int i = -1; i <= 1; ++i) {
			for (int j = -1; j <= 1; ++j) {
				if (i == 0 && j == 0) continue;
				int nx = x + i;
				int ny = y + j;
				// Boundary condition handling
				if (nx >= 0 && nx < N && ny >= 0 && ny < N) {
					//printf("_live_neigh_-%d,%d,%d-_\n", idx, nx, ny);
					live_neighbors += d_grid[ny * N + nx];
					
				}
				
			}
		}

		// Apply the Game of Life rules
		if (d_grid[idx] == 1) {  // Cell is alive
			if (live_neighbors >= 4 || live_neighbors <= 1) {
				d_new_grid[idx] = 0;  // Cell dies
			} else {
				d_new_grid[idx] = 1;  // Cell survives
			}
		} else {  // Cell is dead
			if (live_neighbors == 3) {
				d_new_grid[idx] = 1;  // Cell becomes alive
			} else {
				d_new_grid[idx] = 0;  // Cell remains dead
			}
		}
	}
}

//update alive count e consecutive alive count 
__global__ void update_counts(int *d_grid, int *d_new_grid, int *d_alive_count, int *d_consecutive_alive_count, int N) {

    int x = blockIdx.x * blockDim.x + threadIdx.x; // column
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    int idx = y * N + x; // cell index
	
	if (x< N && y < N){
		// Update the counts based on the new grid state
		if (d_new_grid[idx] == 1) {
			d_alive_count[idx]++;  // Increment total alive count
			
			if (d_grid[idx] == 1){
				d_consecutive_alive_count[idx]++;  // Increment consecutive alive count
			}
		}

		// Update the grid for the next iteration
		d_grid[idx] = d_new_grid[idx];
	}
}

void read_initial_state(const char *filename, int *h_grid, int N) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        perror("Error opening file");
        exit(EXIT_FAILURE);
    }
	
	// read initial state grid
    for (int i = 0; i < N * N; ++i) {
        fscanf(file, "%d", &h_grid[i]);
    }

    fclose(file);
}


void save_results(const char *filename, int N, double execution_time, int blockDim, int gridDim, int generations) {
    FILE *file = fopen(filename, "a"); // Open file in append mode 
    if (file == NULL) {
		FILE *file = fopen(filename, "w"); // Open file in write mode (create file)
			if (file == NULL) {
			perror("Failed to open file");
			exit(EXIT_FAILURE);
			}
    }

    // Write statistics and variables to the file on a single line
    fprintf(file, "%d %.3f %d %d %d\n",
            N, execution_time, blockDim, gridDim, generations);

    fclose(file); // Close the file
}

// Function to compare final CPU and GPU results
int compare_final_results(int *cpu_grid, int *h_grid, int *h_alive_count, int *h_consecutive_alive_count, int N, int generations) {
    
    int *cpu_new_grid = (int *)malloc(N * N * sizeof(int));
    int *cpu_alive_count = (int *)malloc(N * N * sizeof(int));
    int *cpu_consecutive_alive_count = (int *)malloc(N * N * sizeof(int));
	int check_bool=0;
    // Initialize CPU grid and counts with the initial state
    memcpy(cpu_alive_count, cpu_grid, N * N * sizeof(int));
    memset(cpu_consecutive_alive_count, 0, N * N * sizeof(int));

    // Run the game of life on CPU
    for (int gen = 0; gen < generations; ++gen) {
        for (int y = 0; y < N; ++y) {
            for (int x = 0; x < N; ++x) {
				// initialize neighbors counter
                int live_neighbors = 0;

                // Count live neighbors
                for (int i = -1; i <= 1; ++i) {
                    for (int j = -1; j <= 1; ++j) {
                        if (i == 0 && j == 0) continue;
                        int nx = x + j, ny = y + i;
                        if (nx >= 0 && nx < N && ny >= 0 && ny < N) {
                            live_neighbors += cpu_grid[ny * N + nx];
                        }
                    }
                }

                int idx = y * N + x;
                if (cpu_grid[idx] == 1) {
                    cpu_new_grid[idx] = (live_neighbors == 2 || live_neighbors == 3) ? 1 : 0;
                } else {
                    cpu_new_grid[idx] = (live_neighbors == 3) ? 1 : 0;
                }

                // Update alive counts
                cpu_alive_count[idx] += cpu_new_grid[idx];
				if(cpu_new_grid[idx] == 1 && cpu_grid[idx]){cpu_consecutive_alive_count[idx]++; }
            }
        }

        // Swap grids for the next generation
        int *temp = cpu_grid;
        cpu_grid = cpu_new_grid;
        cpu_new_grid = temp;
    }

    // Compare the final CPU and GPU results
    if (memcmp(cpu_grid, h_grid, N * N * sizeof(int)) != 0) {
        printf("Mismatch found in grid values!\n");
		check_bool=1;
    }
    if (memcmp(cpu_alive_count, h_alive_count, N * N * sizeof(int)) != 0) {
        printf("Mismatch found in alive count values!\n");
		check_bool=1;
    }
    if (memcmp(cpu_consecutive_alive_count, h_consecutive_alive_count, N * N * sizeof(int)) != 0) {
        printf("Mismatch found in consecutive alive count values!\n");
		check_bool=1;
    }
    
    free(cpu_new_grid);
    free(cpu_alive_count); 
    free(cpu_consecutive_alive_count);
   
    return check_bool;
}


// MAIN
int main(int argc, char *argv[]) {
    if (argc == 0) {
        fprintf(stderr, "Input Error - Usage: game_of_life.exe <initial_state_file> <N> <blockDim> <generations>\n", argv[0]);
        return 1;
    }
	
	// read input parameter
    const char *filename = argv[1];
    int N = atoi(argv[2]);
    int blockDimX = atoi(argv[3]);
    int blockDimY = blockDimX;    //atoi(argv[4]);
	if( blockDimX * blockDimY > 1024){
	printf("Error: too much threads in a block (>1024)");
	return 1;
	}
    int generations = atoi(argv[4]);
	
	//allocate grids
    size_t grid_size = N * N * sizeof(int);
    int *h_grid = (int *)malloc(grid_size);
    int *h_alive_count = (int *)malloc(grid_size);
    int *h_consecutive_alive_count = (int *)malloc(grid_size);

    read_initial_state(filename, h_grid, N);

	// Initialize counts
    for (int y = 0; y < N; ++y) {
        for (int x = 0; x < N; ++x) {
            h_alive_count[y * N + x] = h_grid[y * N + x];// cell initially alive 
			h_consecutive_alive_count[y * N + x] = 0;  // No consecutive counts initially
        }
    }	

    // Initialize CPU grid and counts with the initial state
    int *cpu_grid = (int *)malloc(N * N * sizeof(int));
    memcpy(cpu_grid, h_grid, N * N * sizeof(int));
	
	//allocate device grids
    int *d_grid, *d_new_grid, *d_alive_count, *d_consecutive_alive_count;
    gpuErrchk(hipMalloc(&d_grid, grid_size));
    gpuErrchk(hipMalloc(&d_new_grid, grid_size));
    gpuErrchk(hipMalloc(&d_alive_count, grid_size));
    gpuErrchk(hipMalloc(&d_consecutive_alive_count, grid_size));

    gpuErrchk(hipMemcpy(d_grid, h_grid, grid_size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_alive_count, h_alive_count, grid_size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_consecutive_alive_count, h_consecutive_alive_count, grid_size, hipMemcpyHostToDevice));

	// CUDA Block Grid dimension 
    dim3 blockDim(blockDimX, blockDimY);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

	printf("CUDA gridDim:= %d x %d",(N + blockDim.x - 1) / blockDim.x  ,(N + blockDim.y - 1) / blockDim.y );
	printf("\n");
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Run the kernel for the specified number of generations
    for (int gen = 0; gen < generations; ++gen) {
        compute_next_generation<<<gridDim, blockDim>>>(d_grid, d_new_grid, N);

        // Synchronize the kernel to ensure all blocks are done
        hipDeviceSynchronize();

        // Update the counts after the new generation is computed
        update_counts<<<gridDim, blockDim>>>(d_grid, d_new_grid, d_alive_count, d_consecutive_alive_count, N);

        // Synchronize the kernel again to ensure the counts are updated before the next generation
        hipDeviceSynchronize();
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Time spent on (BASIC) computation of %d x %d grid: %f ms\n", N, N, milliseconds);
	
	// copy result to host
    hipMemcpy(h_grid, d_grid, grid_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_alive_count, d_alive_count, grid_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_consecutive_alive_count, d_consecutive_alive_count, grid_size, hipMemcpyDeviceToHost);
	
	// check with cpu
	// Check for "--check" flag
    int check = 0;
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--check") == 0) {
            check = 1;
            break;
        }
    }
	if(check){
	
	int check_bool = compare_final_results(cpu_grid,h_grid,h_alive_count,h_consecutive_alive_count, N, generations);
	
		if(!check_bool){ 
			printf("Check result OK \n");
			}
		else{
			printf("Check result Failed \n");
			}
	}
	
	// Check for "--verbose" flag
    int verbose = 0;
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--verbose") == 0) {
            verbose = 1;
            break;
        }
    }
	// If verbose flag is set, write the final result
    if (verbose) {
		// Output file
		FILE *output_file = fopen("result.txt", "w");
		if (output_file == NULL) {
			perror("Error opening result.txt");
			return 1;
		}
	

		// Write the final grid state to the file
		fprintf(output_file, "Final grid state:\n");
		for (int y = 0; y < N; ++y) {
			for (int x = 0; x < N; ++x) {
				fprintf(output_file, "%c ", h_grid[y * N + x] ? 'x' : '_');
			}
			fprintf(output_file, "\n");
		}

		// Write the cell statistics to the file
		fprintf(output_file, "Cell statistics:\n");
		for (int y = 0; y < N; ++y) {
			for (int x = 0; x < N; ++x) {
				int idx = y * N + x;
				fprintf(output_file, "Cell (%d,%d): Total Alive: %d, Consecutive Alive: %d\n", x, y, h_alive_count[idx], h_consecutive_alive_count[idx]);
			}
		}

		// Close the output file
		fclose(output_file);
	}
	
	save_results("data.txt", N, milliseconds, blockDimX, gridDim.x, generations);
	
	//free memory
    hipFree(d_grid);
    hipFree(d_new_grid);
    hipFree(d_alive_count);
    hipFree(d_consecutive_alive_count);
    free(cpu_grid);
    free(h_grid);
    free(h_alive_count);
    free(h_consecutive_alive_count);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
